

#include "dnndiff/ops.h"
#include "dnndiff/cuda_helper.h"
using namespace dnndiff;

void Squeeze::map(void)
{
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Squeeze::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Squeeze::forward(bool block)
{
  copy_kernel<<<GET_BLOCKS(outputs[0].volume()), CUDA_NUM_THREADS>>>(
      (float*)outputs[0].data_ptr, (float*)inputs[0].data_ptr, outputs[0].volume());
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_squeeze_cost(Squeeze* sqz)
{
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    copy_kernel<<<GET_BLOCKS(sqz->outputs[0].volume()), CUDA_NUM_THREADS>>>(
        outputPtr, inputPtr, sqz->outputs[0].volume());
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  sqz->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Squeeeze]: cost(%.4lf)\n", sqz->runtime);
}
