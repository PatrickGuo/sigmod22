

#include "dnndiff/ops.h"
#include "dnndiff/cuda_helper.h"
using namespace dnndiff;

void Where::map(void)
{
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputs[0].volume() * sizeof(DATATYPE)));
}

void Where::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Where::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_where_cost(Where* where)
{
  where->runtime = 0;
}
