#include "hip/hip_runtime.h"


#include "dnndiff/ops.h"
#include "dnndiff/cuda_helper.h"

using namespace dnndiff;

Model::Model()
: isTraining(false), print_cost(false)
{
  //int* a = (int*) malloc(sizeof(int) * 8);
  checkCUDA(hipSetDevice(0));
  checkCUDNN(hipdnnCreate(&dnn));
  checkCUDA(hipblasCreate(&blas));
  checkCUDA(hipsolverDnCreate(&solver)); // DeCor : svd
  workSpaceSize = WORK_SPACE_SIZE;
  global_unique_id = 100;
  checkCUDA(hipMalloc(&workSpace, workSpaceSize));
  // printf("handle.workSpace = 0x%x\n", workSpace);
  // create all descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&scaleTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // allocate tensors for measuring performance
  checkCUDA(hipMalloc(&inputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&biasPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&outputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&filterPtr, MAX_TENSOR_SIZE));
  // create tensors for batch norm
  checkCUDA(hipMalloc(&scalePtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningVar, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveVar, MAX_TENSOR_SIZE));
  // create cuda events
  checkCUDA(hipEventCreate(&startEvent));
  checkCUDA(hipEventCreate(&endEvent));
}

float Model::measure_oplist_runtime(const std::vector<OpBase*>& opBaseList)
{
  const int num_runs = 100;
  // warmup
  for (int times = 0; times < num_runs; times++)
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  // measure runtime
  // checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int times = 0; times < num_runs; times++) {
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  return milliseconds / num_runs;
}

void* Model::allocate_memory(size_t size, const DATATYPE* data_initial)
{
  void* ptr;
  if (size == 0) {
    // Note: Special value for zero-sized tensor
    ptr = (void*) 0x1;
  } else {
    checkCUDA(hipMalloc(&ptr, size));
  }
  if (data_initial != NULL) {
    checkCUDA(hipMemcpy(ptr, data_initial, size, hipMemcpyDefault));
  }
  return ptr;
}

bool Model::copy_memory(DATATYPE* dst, const DATATYPE* src, size_t size)
{
  checkCUDA(hipMemcpy(dst, src, size, hipMemcpyDefault));
  return true;
}
