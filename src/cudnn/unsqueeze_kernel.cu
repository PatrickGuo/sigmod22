

#include "dnndiff/ops.h"
#include "dnndiff/cuda_helper.h"
using namespace dnndiff;

void Unsqueeze::map(void)
{
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Unsqueeze::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Unsqueeze::forward(bool block)
{
  copy_kernel<<<GET_BLOCKS(outputs[0].volume()), CUDA_NUM_THREADS>>>(
      (float*)outputs[0].data_ptr, (float*)inputs[0].data_ptr, outputs[0].volume());
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_unsqueeze_cost(Unsqueeze* unsqz)
{
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    copy_kernel<<<GET_BLOCKS(unsqz->outputs[0].volume()), CUDA_NUM_THREADS>>>(
        outputPtr, inputPtr, unsqz->outputs[0].volume());
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  unsqz->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Squeeeze]: cost(%.4lf)\n", unsqz->runtime);
}
